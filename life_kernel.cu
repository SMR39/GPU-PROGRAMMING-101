
#include <hip/hip_runtime.h>

__global__ void init_kernel(int * domain, int domain_x)
{
	// Dummy initialization
	/*domain[blockIdx.y * domain_x  + blockIdx.x * blockDim.x +  threadIdx.x]
		= (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3; */

		int iy = blockDim.y * blockIdx.y + threadIdx.y;
		int ix = blockDim.x * blockIdx.x + threadIdx.x;
		int idx = iy * domain_x + ix;

		domain[idx] = (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
		__syncthreads();
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}


// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain,
    int domain_x, int domain_y)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;// computing the y-dimension
    //Shared Memory used by all the threads inside the block
	extern __shared__ int shared_source_domain[];

	for (int i=tx; i<tx+8; i++)
	{
		for (int j=ty; j<ty+8; j++)
		{
			shared_source_domain[i * (domain_x/16) + j] = source_domain[i * domain_x + j];
		}
	}
	__syncthreads();
    // Read cell
    int myself = read_cell(shared_source_domain, tx, ty, 0, 0,
	                       domain_x, domain_y);
    
    // TODO: Read the 8 neighbors and count number of blue and red
	int neighbors=0;
	int red=0, blue=0, blank=0;
	for (int i=-1; i<2; i++)
    {
        for (int j=-1; j<2; j++)
        {
            if ((i !=0) || (j !=0))
            {
                neighbors = read_cell(shared_source_domain, tx, ty, i, j, domain_x, domain_y);

                if (neighbors == 1)
                {
                    red++;
                }
                else if (neighbors == 2)
                {
                    blue++;
                }
				else if (neighbors == 0)
				{
					blank++;
				}
            }
        }
    }
	__syncthreads();
	// TODO: Compute new value

	int all_neighbors = red + blue;
	
	//control flow divergence 
	    
	if ((all_neighbors < 2) || (all_neighbors > 3))
    {
        myself = 0;
    }
    else if ((all_neighbors == 2) || (all_neighbors == 3))
    {
        if ( blue >= 2)
        {
            myself = 2;
        }
        else
        {
            myself = 1;
        }
    }
    __syncthreads();
    // TODO: Write it in dest_domain
    
    dest_domain[(ty * domain_x) + tx] = myself;
}

